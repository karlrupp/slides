#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <vector>

#include "timer.hpp"

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %sn", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}


__global__ void add(double *a, double *b, double *c)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  a[i] = b[i] + c[i];
}



template <typename T>
void runTest(int deviceId, int blocknum)
{
  T *d_a;
  T *d_b;
  T *d_c;

  int N = blocknum*256;

  std::vector<T> vec_init(N);

  // NB:  d_a(33*nMB) for stride case
  checkCuda( hipMalloc(&d_a, N * sizeof(T)) );
  checkCuda( hipMalloc(&d_b, N * sizeof(T)) );
  checkCuda( hipMalloc(&d_c, N * sizeof(T)) );

  checkCuda( hipMemcpy(&d_a, &(vec_init[0]), N * sizeof(T), hipMemcpyHostToDevice) );
  checkCuda( hipMemcpy(&d_b, &(vec_init[0]), N * sizeof(T), hipMemcpyHostToDevice) );
  checkCuda( hipMemcpy(&d_c, &(vec_init[0]), N * sizeof(T), hipMemcpyHostToDevice) );

  add<<<blocknum, 256>>>(d_a, d_b, d_c); // warm up
  hipDeviceSynchronize();

  viennacl::tools::timer timer;
  timer.start();
  for (int n = 0; n < 10; ++n)
  {
    add<<<blocknum, 256>>>(d_a, d_b, d_c);
  }
  hipDeviceSynchronize();
  double bandwidth = 30 * N * sizeof(double) / timer.get() / 1e9;
  std::cout << N << "    " << bandwidth << "     " << timer.get() << std::endl;


  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

int main(int argc, char **argv)
{
  int deviceId = 0;
  hipDeviceProp_t prop;

  checkCuda( hipGetDeviceProperties(&prop, deviceId) );
  std::cout << "Device: " << prop.name << std::endl;
  std::cout << "#Size   Bandwidth (GB/s)    Time (sec)" << std::endl;

  for (int blocks=1; blocks<20000; blocks *= 2)
    runTest<double>(deviceId, blocks);

  return EXIT_SUCCESS;
}
