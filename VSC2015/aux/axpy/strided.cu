#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>

#include "timer.hpp"

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %sn", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}


__global__ void offset(double *a, double *b, int s)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x + s;
  b[i] = a[i] + 1;
}


__global__ void stride(double *a, double *b, int s)
{
  int i = (blockDim.x * blockIdx.x + threadIdx.x) * s;
  b[i] = a[i] + 1;
}



template <typename T>
void runTest(int deviceId)
{
  T *d_a;
  T *d_b;

  int N = 1024*256;

  // NB:  d_a(33*nMB) for stride case
  checkCuda( hipMalloc(&d_a, N * 33 * sizeof(T)) );
  checkCuda( hipMalloc(&d_b, N * 33 * sizeof(T)) );

  offset<<<1024, 256>>>(d_a, d_b, 0); // warm up
  hipDeviceSynchronize();

  std::cout << "#Offset, Bandwidth (GB/s)" << std::endl;

  viennacl::tools::timer timer;
  for (int i = 0; i <= 32; i++) {
    timer.start();
    for (int n = 0; n < 10; ++n)
    {
      offset<<<1024, 256>>>(d_a, d_b, i);
    }
    hipDeviceSynchronize();
    double bandwidth = 20 * 1024 * 256 * sizeof(double) / timer.get() / 1e9;
    std::cout << i << "    " << bandwidth << std::endl;
  }

  std::cout << "#Stride, Bandwidth (GB/s)" << std::endl;

  stride<<<1024, 256>>>(d_a, d_b, 0); // warm up
  hipDeviceSynchronize();

  for (int i = 1; i <= 32; i++) {
    timer.start();
    for (int n = 0; n < 10; ++n)
    {
      stride<<<1024, 256>>>(d_a, d_b, i);
    }
    hipDeviceSynchronize();
    double bandwidth = 20 * 1024 * 256 * sizeof(double) / timer.get() / 1e9;
    std::cout << i << "    " << bandwidth << std::endl;
  }

  hipFree(d_a);
}

int main(int argc, char **argv)
{
  int deviceId = 0;
  hipDeviceProp_t prop;

  checkCuda( hipGetDeviceProperties(&prop, deviceId) );
  std::cout << "Device: " << prop.name << std::endl;

  runTest<double>(deviceId);

  return EXIT_SUCCESS;
}
